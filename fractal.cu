#include "hip/hip_runtime.h"
/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2016, Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is not permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
Co-Author: Darren Rambaud
*/

#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "cs43805351.h"
#include <math.h>

static const int ThreadsPerBlock = 512;

static const double Delta = 0.005491;
static const double xMid = 0.745796;
static const double yMid = 0.105089;

static __global__
void FractalKernel(const int frames, const int width, unsigned char pic[])
{
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < frames * (width * width)) {
        const int col = idx % width;
        const int row = (idx / width) % width;
        const int frame = idx / (width * width);

        const double myDelta = Delta * pow(0.99, frame+1); //loop dep fixed
        const double xMin = xMid - myDelta;
        const double yMin = yMid - myDelta;
        const double dw = 2.0 * myDelta / width;
        //todo: compute a single pixel here
        if (row < width) { // bounds checking, ensures no wasted calc
            const double cy = -yMin - row * dw;
            if (col < width) { // bounds checking
                const double cx = -xMin - col * dw;
                double x = cx;
                double y = cy;
                int depth = 256;
                double x2, y2;
                do {
                    x2 = x * x;
                    y2 = y * y;
                    y = 2 * x * y + cy;
                    x = x2 - y2 + cx;
                    depth--;
                } while ((depth > 0) && ((x2 + y2) < 5.0));
                pic[frame * width * width + row * width + col] \
                = (unsigned char)depth;
            }
        }
    }
}

int main(int argc, char *argv[])
{
    printf("Fractal v1.5 [CUDA]\n");

    // check command line
    if (argc != 3) {
        fprintf(stderr, "usage: %s frame_width num_frames\n", \
                argv[0]); 
        exit(-1);
    }
    int width = atoi(argv[1]);
    if (width < 10) {
        fprintf(stderr, "error: frame_width must be at least 10\n"); 
        exit(-1);
    }
    int frames = atoi(argv[2]);
    if (frames < 1) {
        fprintf(stderr, "error: num_frames must be at least 1\n"); 
        exit(-1);
    }
    printf("computing %d frames of %d by %d fractal\n", \
           frames, width, width);

    // allocate picture array
    unsigned char* pic = new unsigned char[frames * width * width];
    unsigned char* pic_d;
    if (hipSuccess != \
        hipMalloc((void **)&pic_d, frames * width * width * \
         sizeof(unsigned char))) {
        fprintf(stderr, "could not allocate memory\n"); 
        exit(-1);
    }

    // start time
    struct timeval start, end;
    gettimeofday(&start, NULL);

    // compute frames
    FractalKernel<<<\
     (frames * width * width + (ThreadsPerBlock - 1)) / ThreadsPerBlock, \
        ThreadsPerBlock>>>(frames, width, pic_d);
    if (hipSuccess != \
         hipMemcpy(pic, pic_d, frames * width * width * \
          sizeof(unsigned char), hipMemcpyDeviceToHost)) {
        fprintf(stderr, "copying from device failed\n"); 
        exit(-1);
    }

    // end time
    gettimeofday(&end, NULL);
    double runtime = end.tv_sec + end.tv_usec / 1000000.0 - \
                     start.tv_sec - start.tv_usec / 1000000.0;
    printf("compute time: %.4f s\n", runtime);

    // verify result by writing frames to BMP files
    if ((width <= 400) && (frames <= 30)) {
        for (int frame = 0; frame < frames; frame++) {
            char name[32];
            sprintf(name, "fractal%d.bmp", frame + 1000);
            writeBMP(width, width, &pic[frame * width * width], name);
        }
    }

    delete [] pic;
    hipFree(pic_d);
    return 0;
}
